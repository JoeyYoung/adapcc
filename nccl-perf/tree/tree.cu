#include "hip/hip_runtime.h"
/*
    Test for a simple tree, should support GPU in any server
        GPU0 ------ GPU1
                      | 
                ------------
               GPU2       GPU3
    test for the latency of nccl send and recv

*/

#include "nccl.h"
#include "hip/hip_runtime.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

static void getHostName(char *hostname, int maxlen) {
    gethostname(hostname, maxlen);
    for (int i = 0; i < maxlen; i++) {
        if (hostname[i] == '.') {
            hostname[i] = '\0';
            return;
        }
    }
}

static uint64_t getHostHash(const char *string) {
    // Based on DJB2, result = result * 33 + char
    uint64_t result = 5381;
    for (int c = 0; string[c] != '\0'; c++) {
        result = ((result << 5) + result) + string[c];
    }
    return result;
}

void printCudaArray(float* device, int size){
    float* host = (float*)malloc(sizeof(float)*size);
    hipMemcpy(host, device, size*sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < size; i++)   printf("%f ", host[i]);
    free(host);
}

__global__ void reduceKernel(float* a, float* b, float* c, int size) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x){
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char *argv[]){
    int myRank, nRanks, localRank = 0;

    // initializing MPI
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myRank);
    MPI_Comm_size(MPI_COMM_WORLD, &nRanks);

    // obtain local rank
    uint64_t hostHashs[nRanks];
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD);
    for (int p = 0; p < nRanks; p++) {
        if (p == myRank) {
            break;
        }
        if (hostHashs[p] == hostHashs[myRank]) {
            localRank++;    // get local rank index
        }
    }

    // nccl id
    ncclUniqueId id;
    ncclComm_t comm;
    if (myRank == 0) {
        ncclGetUniqueId(&id);   // ncclInit: initEnv() initNet()
    }
    MPI_Bcast((void *) &id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);


    // cuda part: allocate mem
    int size = 32;
    float *sendbuff, *recvbuff;
    hipStream_t s;
    hipSetDevice(localRank);
    hipStreamCreate(&s);

    // nccl init
    ncclCommInitRank(&comm, nRanks, id, myRank);
    
    clock_t start,end;

    start = clock();
    // set reduce tree
    if (myRank == 2 or myRank == 3){
        // send data to GPU 1
        int dst = 1;
        hipMalloc(&sendbuff, size * sizeof(float)); // 4 bytes unit size
        
        // set init value in host mem and copy to device
        float* sendbuff_host = (float*) malloc(sizeof(float) * size);
        for(int i = 0; i < size; i++)   sendbuff_host[i] = float(myRank);
        hipMemcpy(sendbuff, sendbuff_host, size*sizeof(float), hipMemcpyHostToDevice);
        free(sendbuff_host);
        
        ncclSend(sendbuff, size, ncclFloat, dst, comm, s);

        hipStreamSynchronize(s);
    }else if(myRank == 1){
        // recv data from GPU 2 and GPU 3
        int dst = 0;
        int peer2 = 2;
        int peer3 = 3;
        hipMalloc(&recvbuff, 2 * size * sizeof(float));
        hipMalloc(&sendbuff, size * sizeof(float));
        
        ncclGroupStart();
        ncclRecv(recvbuff, size, ncclFloat, peer2, comm, s);
        ncclRecv(recvbuff + size, size, ncclFloat, peer3, comm, s);
        ncclGroupEnd();

        // reduce operation, set send buffer, wait for nccl recv?
        int nblocks = 2;
        reduceKernel <<<nblocks, size/nblocks>>> (recvbuff, recvbuff + size, sendbuff, size);
        hipDeviceSynchronize();
        
        ncclSend(sendbuff, size, ncclFloat, dst, comm, s);

        hipStreamSynchronize(s);
    }else if(myRank == 0){
        // recv data from GPU 1
        int src = 1;
        hipMalloc(&recvbuff, size * sizeof(float));
        sleep(3);
        ncclRecv(recvbuff, size, ncclFloat, src, comm, s);
        
        hipStreamSynchronize(s);
    }
    end = clock();
    printf("time=%f(ms)\n",(double)(end-start)/CLOCKS_PER_SEC * 1000);  

    // free device buffers, finalizing NCCL, MPI
    hipFree(sendbuff);
    hipFree(recvbuff);
    ncclCommDestroy(comm);
    MPI_Finalize();

    printf("[Tree MPI Rank %d] Success \n", myRank);
    return 0;
}
