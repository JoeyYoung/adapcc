/*
    Reduce Context
    - multiple parallel transmission contexts
*/

#include "control.h"
#include "trans.h"
#include "time.h"
#include <vector>
#include <iostream>
using namespace std;

static std::queue<workElement*> workQueueReduce[MAX_TRANS];
static std::queue<resultElement*> resultQueue[MAX_TRANS];

static MPIInfo* myMPIInfo;
static int numTrans;
static RankRoleInfo roles[MAX_TRANS][MAX_DEVICES];

static char* ipTable[MAX_DEVICES];
static vector<int> localMasterList;
static pthread_t pidsReduce[MAX_TRANS];

static int initCount;
static pthread_mutex_t initCountMutex;
static bool initDone;

static int workCount;
static pthread_mutex_t workCountMutex;
static bool workDone;

static socketFds* globalIpcFds;
static socketFds* workIpcFds;

static bool exitSignal;

static void treeDFS(XMLElement* node, int parent, int tid){
    int id = atoi(node->Attribute("id"));
    char* ip = (char*)node->Attribute("ip");
    ipTable[id] = ip;

    roles[tid][id].siblingIdx = 0;
    XMLElement* tmp = node->NextSiblingElement("gpu");
    while(tmp != NULL){
        tmp = tmp->NextSiblingElement("gpu");
        roles[tid][id].siblingIdx += 1;
    }
    if(parent != -1){
        roles[tid][id].subsequents.insert(parent);
    }
    XMLElement* child = node->FirstChildElement("gpu");
    while(child != NULL){
        int childId = atoi(child->Attribute("id"));
        roles[tid][id].precedents.insert(childId);
        child = child->NextSiblingElement("gpu");
    }

    if(node->FirstChildElement("gpu") == NULL)
        return;
    
    child = node->FirstChildElement("gpu");
    while(child != NULL){
        treeDFS(child, id, tid);
        child = child->NextSiblingElement();
    }
}

static void getStrategyFromXML(XMLElement* trees){
    XMLElement *root = trees->FirstChildElement("root");
    int count = 0;
    while(root != NULL){
        treeDFS(root, -1, count);
        count += 1;
        root = root->NextSiblingElement("root");
    }
    numTrans = count;

    char temp_ip[20];
    localMasterList.clear();
    for(int i = 0; i < myMPIInfo->nRanks; i++){
        if(strcmp(ipTable[i], temp_ip) != 0){
            strcpy(temp_ip, ipTable[i]);
            localMasterList.push_back(i);
        }
    }
}

static void globalBarrier(){
    char initBarrier = 'I';
    for(int i = 0; i < myMPIInfo->nRanks; i++){
        if(i == myMPIInfo->myRank) continue;
        socketIpcSend(globalIpcFds->sendFd[i], &initBarrier, 1);
    }
    for(int i = 0; i < myMPIInfo->nRanks; i++){
        if(i == myMPIInfo->myRank) continue;
        socketIpcRecv(globalIpcFds->recvFd[i], &initBarrier, 1);
    }
}

static void globalInitBarrier(int tid){
    char initBarrier = 'I';
    pthread_mutex_lock(&initCountMutex);
    initCount += 1;
    pthread_mutex_unlock(&initCountMutex);
    
    if(tid != 0)
        while(!initDone);
    else{
        while(initCount == 0 || initCount%numTrans!=0);
        for(int i = 0; i < myMPIInfo->nRanks; i++){
            if(i == myMPIInfo->myRank) continue;
            socketIpcSend(globalIpcFds->sendFd[i], &initBarrier, 1);
        }
        for(int i = 0; i < myMPIInfo->nRanks; i++){
            if(i == myMPIInfo->myRank) continue;
            socketIpcRecv(globalIpcFds->recvFd[i], &initBarrier, 1);
        }
        initDone = true;
    }
}

static void workElemBarrier(int tid, bool* records){
    char workBarrier = 'W';
    pthread_mutex_lock(&workCountMutex);
    workCount += 1;
    pthread_mutex_unlock(&workCountMutex);

    if(tid != 0){
        while(!workDone);
    }
    else{
        while(workCount==0 || workCount%(numTrans)!=0);
        for(int i = 0; i < myMPIInfo->nRanks; i++){
            if(i == myMPIInfo->myRank) continue;
            socketIpcSend(workIpcFds->sendFd[i], &workBarrier, 1);
        }
        for(int i = 0; i < myMPIInfo->nRanks; i++){
            if(i == myMPIInfo->myRank) continue;
            socketIpcRecv(workIpcFds->recvFd[i], &workBarrier, 1);
        }
        workDone = true;
    }

    if(myMPIInfo->localRank == 0)
        memset(records, 0, myMPIInfo->nRanks*MAX_CHUNK_NUM);
    globalInitBarrier(tid);
}

static void workElemBarrierReb(){
    workDone = false;
}

static void* reduceThreadFunc(void* args){
    hipSetDevice(myMPIInfo->localRank);
    hipDeviceEnablePeerAccess(myMPIInfo->localRank, 0);
    int tid = *(int*)args;
    hipStream_t streamR;
    hipStreamCreate(&streamR);

    RankRoleInfo role = roles[tid][myMPIInfo->myRank];
    float* recvDevBuffer;
    hipMalloc(&recvDevBuffer, sizeof(float)*MAX_BUF_SIZE*role.precedents.size());
    
    sharedMemoryInfo shm_info;
    volatile shmStruct* shm;

    char shm_name[] = "shm_nameR_";
    shm_name[10] = '0' + tid;
    sharedMemoryCreate(shm_name, sizeof(*shm), &shm_info);
    shm = (volatile shmStruct *)shm_info.addr;
    hipIpcGetMemHandle(
        (hipIpcMemHandle_t *)&shm->memHandle[myMPIInfo->myRank], recvDevBuffer
    );

    for(int i=0; i<myMPIInfo->nRanks; i++){
        for(int j=0; j<MAX_CHUNK_NUM; j++){
            hipEvent_t event;
            hipEventCreate(&event, hipEventDisableTiming | hipEventInterprocess);
            hipIpcGetEventHandle(
                (hipIpcEventHandle_t *)&shm->eventHandle[i][j], event
            );
        }
    }

    bool* eventRecords; 
    int shmID = shmEventRecordsCreate(&eventRecords, tid, 0, 
                    myMPIInfo->myRank, myMPIInfo->nRanks);
    
    globalInitBarrier(tid);
    printf("[Rank %d]global init done.\n", myMPIInfo->myRank);
    void* ptrs[myMPIInfo->nRanks];
    hipEvent_t ipcEvents[myMPIInfo->nRanks][MAX_CHUNK_NUM];
    set<int>::iterator it;

    for(it=role.subsequents.begin(); it!=role.subsequents.end(); it++){
        int dstRank = *it;
        void* p;
        hipIpcOpenMemHandle(
            &p, *(hipIpcMemHandle_t *)&shm->memHandle[dstRank], 
            hipIpcMemLazyEnablePeerAccess
        );
        ptrs[dstRank] = p;
    }

    // pre-load event ipc
    for(int i=0; i<myMPIInfo->nRanks; i++){
        for(int cid=0; cid<MAX_CHUNK_NUM; cid++){
            hipEvent_t p;
            hipIpcOpenEventHandle(&p, 
                *(hipIpcEventHandle_t *)&shm->eventHandle[i][cid]);
            ipcEvents[i][cid] = p;
        }
    }
    unordered_map<int, bool> checkCrossNodeSend;
    unordered_map<int, bool> checkCrossNodeRecv;
    unordered_map<int, int> checkPreSiblingIdx;

    for(it=role.subsequents.begin(); it!=role.subsequents.end(); it++){
        int dstRank = *it;
        if(strcmp(ipTable[myMPIInfo->myRank], ipTable[dstRank]) != 0){
            printf("[Rank %d]Reduce Thread: Cross send to %d\n", 
                    myMPIInfo->myRank, dstRank);
            checkCrossNodeSend[dstRank] = true;
        }
    }

    int preCount = 0;
    // cross node receive check
    for(it=role.precedents.begin(); it!=role.precedents.end(); it++){
        int srcRank = *it;
        if(strcmp(ipTable[myMPIInfo->myRank], ipTable[srcRank]) != 0){
            printf("[Rank %d]Reduce Thread: Cross receive from %d\n", 
                    myMPIInfo->myRank, srcRank);
            checkCrossNodeRecv[srcRank] = true;
        }
        preCount += 1;
        checkPreSiblingIdx[srcRank] = role.precedents.size()-preCount;
    }

    relayController* controller = (relayController*)malloc(sizeof(struct relayController_st));
    while(!exitSignal){
        if(workQueueReduce[tid].size() != 0){
            workElemBarrier(tid, eventRecords);

            workElement* elem = workQueueReduce[tid].front();
            resultElement* resElem = resultQueue[tid].front();

            int tranSize = elem->size/numTrans; 
            int tranChunkNum = sizeof(float)*tranSize/elem->chunkBytes; 
            int chunkFloatNum = elem->chunkBytes/sizeof(float); 
            int localBufOffset = tid*tranSize;

            printf("[Rank %d]fetch: address %p\n", myMPIInfo->myRank, elem->tensorBuf);
            float* localTensorDevBuffer = elem->tensorBuf;

            setRelayController(controller, roles[tid], myMPIInfo->myRank, 'r');
            set<int> recvSet = getActiveRecvs(roles[tid], myMPIInfo->myRank, 'r');
            if(role.subsequents.size() == 0){
                if(controller->hasKernel || !controller->hasRecv){  
                    resElem->resultDevBuf = localTensorDevBuffer+localBufOffset;
                }else{
                    if(recvSet.size() != 1) 
                        printf("[Rank %d]Conflict in root<->bcst control.\n", myMPIInfo->myRank);
                    resElem->resultDevBuf = recvDevBuffer +
                                            MAX_BUF_SIZE * checkPreSiblingIdx[*recvSet.begin()];
                }
            }else{
                resElem->resultDevBuf = localTensorDevBuffer+localBufOffset;
            }

            char signal = 'C';
            if(role.precedents.size() == 0 && role.subsequents.size() == 0)
                return 0;
            for(int cid = 0; cid < tranChunkNum; cid++){
                int globalChunkID = tid*tranChunkNum+cid;
                if(controller->hasRecv){
                    for(it=role.precedents.begin(); it!=role.precedents.end(); it++){
                        int srcRank = *it;
                        if(recvSet.count(srcRank) == 0)
                            continue;

                        int recvOffset = MAX_BUF_SIZE * checkPreSiblingIdx[srcRank]
                                            + cid * chunkFloatNum;
                        bool isCrossNode = (checkCrossNodeRecv.count(srcRank) != 0);
                        cudaRecv(
                            srcRank,
                            recvDevBuffer,
                            recvOffset,
                            chunkFloatNum,
                            &signal,
                            globalChunkID,
                            isCrossNode,
                            streamR,
                            ipcEvents[srcRank][globalChunkID],
                            eventRecords,
                            shmEventRecordsIdx(srcRank, globalChunkID)
                        );
                    }
                }

                if(controller->hasKernel){
                    int nBlocks = NUM_KERN_BLOCK;
                    int nThreads = NUM_KERN_THREADS;
                    int elemNum = role.precedents.size();

                    reduceSumKernel <<<nBlocks, nThreads, 0, streamR>>>(
                        recvDevBuffer, 
                        localTensorDevBuffer+localBufOffset+cid*chunkFloatNum,
                        cid, 
                        chunkFloatNum, 
                        MAX_BUF_SIZE, 
                        elemNum, 
                        controller->hasLocal
                    );
                    hipStreamSynchronize(streamR);
                }

                if(controller->hasSend){
                    float* sendBuf = localTensorDevBuffer;
                    int sendOffset = localBufOffset+cid*chunkFloatNum;
                    if(controller->hasRecv && !controller->hasKernel){
                        if(recvSet.size() != 1) 
                            printf("[Rank %d]Conflict in send control.\n", myMPIInfo->myRank);
                        sendBuf = recvDevBuffer;
                        sendOffset = MAX_BUF_SIZE*checkPreSiblingIdx[*recvSet.begin()]
                                        + cid * chunkFloatNum;
                    }

                    for(it=role.subsequents.begin(); it!=role.subsequents.end(); it++){
                        int dstRank = *it;
                        void* ptr = ptrs[dstRank];
                        int startOffset = role.siblingIdx*MAX_BUF_SIZE*sizeof(float);            
                        bool isCrossNode = (checkCrossNodeSend.count(dstRank) != 0);
                        bool setRecord = (*it==*(role.subsequents.rbegin()) ? true: false);
                        cudaSend(
                            myMPIInfo->myRank, 
                            dstRank, 
                            myMPIInfo->localRank,
                            ptr,
                            startOffset+cid*elem->chunkBytes,
                            sendBuf,
                            sendOffset,
                            chunkFloatNum,
                            streamR,
                            &signal,
                            globalChunkID,
                            isCrossNode,
                            ipcEvents[myMPIInfo->myRank][globalChunkID],
                            eventRecords,
                            shmEventRecordsIdx(myMPIInfo->myRank, globalChunkID), 
                            setRecord
                        );
                    }
                }
            }
            resElem->status = true;
            printf("[Rank %d]work elem done.\n", myMPIInfo->myRank);
            workElemBarrierReb();
            workQueueReduce[tid].pop();
        }
    }

    sharedMemoryClose(&shm_info);
    shmEventRecordsRemove(&eventRecords, shmID);
    hipFree(recvDevBuffer);
    return 0;
}

static bool checkResQueue(){
    for(int tid = 0; tid < numTrans; tid++){
        if(!resultQueue[tid].front()->status)
            return false;
    }
    return true;
}

/* API */
namespace reduceContext{

void getMPIInfo(MPIInfo* info){
    myMPIInfo = info;
}

float** processWorkElem(workElement* elem){
    initDone = false;
    globalBarrier();
    printf("[Rank %d]enqueue: %p\n", myMPIInfo->myRank, elem->tensorBuf);

    struct timespec start = {0, 0};
    struct timespec end = {0, 0};
    clock_gettime(CLOCK_THREAD_CPUTIME_ID, &start);

    resultElement* res[numTrans];
    for(int tid = 0; tid < numTrans; tid++){
        res[tid] = (struct resultElement_st*)malloc(sizeof(struct resultElement_st));
        resultQueue[tid].push(res[tid]);
    }

    for(int tid = 0; tid < numTrans; tid++){
        workQueueReduce[tid].push(elem);
    }

    while(!checkResQueue());
    float** returnBufs = (float**)malloc(numTrans * sizeof(float*));
    for(int tid = 0; tid < numTrans; tid++){
        returnBufs[tid] = resultQueue[tid].front()->resultDevBuf;
        resultQueue[tid].pop();
        free(res[tid]);
    }
    clock_gettime(CLOCK_THREAD_CPUTIME_ID, &end);
    double duration = (end.tv_sec - start.tv_sec) * 1000 
                        + (end.tv_nsec - start.tv_nsec) / 1000000;
    globalBarrier();
    return returnBufs;
}

void bootstrapFromXMl(char* filename){
    XMLDocument xml;
    xml.LoadFile(filename);
    XMLElement *trees = xml.RootElement();

    getStrategyFromXML(trees);
    socketFds** channels = buildIpcChannels(ipTable, 
                            myMPIInfo->myRank, myMPIInfo->nRanks);
    globalIpcFds = channels[0];
    workIpcFds = channels[1];

    int threadArgs[numTrans];
    for(int tid = 0; tid < numTrans; tid ++){
        threadArgs[tid] = tid;
        pthread_create(&pidsReduce[tid], NULL, 
            reduceThreadFunc, (void *)&(threadArgs[tid]));
    }
}

void clear(){
    void* status;
    for(int tid = 0; tid < numTrans; tid++){
        pthread_join(pidsReduce[tid], &status);
    }
}

void terminate(){
    exitSignal = true;
}

int getNumTrans(){
    return numTrans;
}

} // end namespace
